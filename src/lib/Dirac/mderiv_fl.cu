/*
 *
 Copyright (C) 2006-2008 Sarod Yatawatta <sarod@users.sf.net>  
 This program is free software; you can redistribute it and/or modify
 it under the terms of the GNU General Public License as published by
 the Free Software Foundation; either version 2 of the License, or
 (at your option) any later version.
 
 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.
 
 You should have received a copy of the GNU General Public License
 along with this program; if not, write to the Free Software
 Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA
 $Id$
 */

#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <stdio.h>

/* enable this for checking for kernel failure */
//#define CUDA_DBG

__global__ void kernel_diagdiv_fl(int M, float eps, float *y, float *x){
  unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
  /* make sure to use only M threads */
  if (tid<M) {
    if (x[tid]>eps) {
      y[tid]=y[tid]/x[tid];
    } else {
      y[tid]=0.0f;
    }
  }
}

__global__ void kernel_diagmu_fl(int M, float *A,float mu){
  unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
  /* make sure to use only M threads */
  if (tid<M) {
    A[tid*(M+1)]=A[tid*(M+1)]+mu;
  } 
}


__global__ void kernel_func_fl(int Nbase, float *x, float *coh, float *p, short *bb, int N){
  /* global thread index : equal to the baseline */
  unsigned int n = threadIdx.x + blockDim.x*blockIdx.x;

  /* this thread works on 
    x[8*n:8*n+7], coh[8*M*n:8*M*n+8*M-1]
    bb[2*n:2*n+1] (sta1,sta2)
    organization of p (N stations and M clusters)
             sta 0          sta 1           sta 2        ....  sta N-1 
  clus 0   0...7            8...15          16...23      ...   8N-8     8N-1
  clus 1   8N..8N+7         8N+8..8N+15     8N+16..8N+23 ....  8N+8N-8...8N+8N-1
  ......
  clus M-1 (M-1)N..(M-1)N+7 (M-1)N+8..(M-1)N+15....  ...(M-1)N+8N-8 (M-1)N+8N-1

    organization of coherencies (coh)
        [0, 8*M-1] : baseline 0
        [8*M, 8*M+8*M-1]: baseline 1
        [n*8*M, n*8*M+8*M-1]: baseline n
        ......
        [n*8*M+cm*8, n*8*M+cm*8+7]  cluster cm, baseline n

    residual error stored at sum[n]
  */ 

  if(n<Nbase) {
    int sta1=(int)bb[2*n];
    int sta2=(int)bb[2*n+1];

    /* condition for calculating this baseline sum is 
      1) its not flagged (sta1,sta2)>=0
    */
    if (sta1>=0 && sta2>=0) {   
     hipFloatComplex G1[4];
     float pp[8]; 
     pp[0]=p[sta1*8];
     pp[1]=p[sta1*8+1];
     pp[2]=p[sta1*8+2];
     pp[3]=p[sta1*8+3];
     pp[4]=p[sta1*8+4];
     pp[5]=p[sta1*8+5];
     pp[6]=p[sta1*8+6];
     pp[7]=p[sta1*8+7];
     G1[0].x=pp[0];
     G1[0].y=pp[1];
     G1[1].x=pp[2];
     G1[1].y=pp[3];
     G1[2].x=pp[4];
     G1[2].y=pp[5];
     G1[3].x=pp[6];
     G1[3].y=pp[7];
     

     hipFloatComplex C[4];
     C[0].x=coh[8*n];
     C[0].y=coh[8*n+1];
     C[1].x=coh[8*n+2];
     C[1].y=coh[8*n+3];
     C[2].x=coh[8*n+4];
     C[2].y=coh[8*n+5];
     C[3].x=coh[8*n+6];
     C[3].y=coh[8*n+7]; 
 
     hipFloatComplex T1[4];
     /* T=G1*C */
     T1[0]=hipCaddf(hipCmulf(G1[0],C[0]),hipCmulf(G1[1],C[2]));
     T1[1]=hipCaddf(hipCmulf(G1[0],C[1]),hipCmulf(G1[1],C[3]));
     T1[2]=hipCaddf(hipCmulf(G1[2],C[0]),hipCmulf(G1[3],C[2]));
     T1[3]=hipCaddf(hipCmulf(G1[2],C[1]),hipCmulf(G1[3],C[3]));

     hipFloatComplex G2[4];
     /* conjugate this */
     pp[0]=p[sta2*8];
     pp[1]=-p[sta2*8+1];
     pp[2]=p[sta2*8+2];
     pp[3]=-p[sta2*8+3];
     pp[4]=p[sta2*8+4];
     pp[5]=-p[sta2*8+5];
     pp[6]=p[sta2*8+6];
     pp[7]=-p[sta2*8+7];
     G2[0].x=pp[0];
     G2[0].y=pp[1];
     G2[2].x=pp[2];
     G2[2].y=pp[3];
     G2[1].x=pp[4];
     G2[1].y=pp[5];
     G2[3].x=pp[6];
     G2[3].y=pp[7];

     hipFloatComplex T2[4];
     T2[0]=hipCaddf(hipCmulf(T1[0],G2[0]),hipCmulf(T1[1],G2[2]));
     T2[1]=hipCaddf(hipCmulf(T1[0],G2[1]),hipCmulf(T1[1],G2[3]));
     T2[2]=hipCaddf(hipCmulf(T1[2],G2[0]),hipCmulf(T1[3],G2[2]));
     T2[3]=hipCaddf(hipCmulf(T1[2],G2[1]),hipCmulf(T1[3],G2[3]));
     /* update model vector */
     x[8*n]=T2[0].x;
     x[8*n+1]=T2[0].y;
     x[8*n+2]=T2[1].x;
     x[8*n+3]=T2[1].y;
     x[8*n+4]=T2[2].x;
     x[8*n+5]=T2[2].y;
     x[8*n+6]=T2[3].x;
     x[8*n+7]=T2[3].y;

    } 
   }

}

__global__ void kernel_jacf_fl(int Nbase, int M, float *jac, float *coh, float *p, short *bb, int N){
  /* global thread index : equal to the baseline */
  unsigned int n = threadIdx.x + blockDim.x*blockIdx.x;
  /* which parameter:0...M */
  unsigned int m = threadIdx.y + blockDim.y*blockIdx.y;

  /* this thread works on 
    x[8*n:8*n+7], coh[8*M*n:8*M*n+8*M-1]
    bb[2*n:2*n+1] (sta1,sta2)
    organization of p (N stations and M clusters)
             sta 0          sta 1           sta 2        ....  sta N-1 
  clus 0   0...7            8...15          16...23      ...   8N-8     8N-1
  clus 1   8N..8N+7         8N+8..8N+15     8N+16..8N+23 ....  8N+8N-8...8N+8N-1
  ......
  clus M-1 (M-1)N..(M-1)N+7 (M-1)N+8..(M-1)N+15....  ...(M-1)N+8N-8 (M-1)N+8N-1

    organization of coherencies (coh)
        [0, 8*M-1] : baseline 0
        [8*M, 8*M+8*M-1]: baseline 1
        [n*8*M, n*8*M+8*M-1]: baseline n
        ......
        [n*8*M+cm*8, n*8*M+cm*8+7]  cluster cm, baseline n

    residual error stored at sum[n]
  */ 

  if(n<Nbase && m<M) {
    int sta1=(int)bb[2*n];
    int sta2=(int)bb[2*n+1];
    /* condition for calculating this baseline sum is 
     If this baseline is flagged,
     or if this parameter does not belong to sta1 or sta2
     we do not compute
    */
    //int stc=m/8; /* 0...Ns-1 (because M=total par= 8 * Nstations */
    int stc=m>>3; /* 0...Ns-1 (because M=total par= 8 * Nstations */

    if (((stc==sta2)||(stc==sta1)) && sta1>=0 && sta2>=0 ) {   

     hipFloatComplex C[4];
     C[0].x=coh[8*n];
     C[0].y=coh[8*n+1];
     C[1].x=coh[8*n+2];
     C[1].y=coh[8*n+3];
     C[2].x=coh[8*n+4];
     C[2].y=coh[8*n+5];
     C[3].x=coh[8*n+6];
     C[3].y=coh[8*n+7]; 
 
     /* which parameter exactly 0..7 */
     //int stoff=m%8;
     int stoff=m-stc*8;
     float pp1[8]; 
     float pp2[8]; 
     if (stc==sta1) {
      for (int cn=0; cn<8; cn++) {
       pp1[cn]=0.0f;
       pp2[cn]=p[sta2*8+cn];
      }
      pp1[stoff]=1.0f;
     } else if (stc==sta2) {
      for (int cn=0; cn<8; cn++) {
       pp2[cn]=0.0f;
       pp1[cn]=p[sta1*8+cn];
      }
      pp2[stoff]=1.0f;
     }


     hipFloatComplex G1[4];
     G1[0].x=pp1[0];
     G1[0].y=pp1[1];
     G1[1].x=pp1[2];
     G1[1].y=pp1[3];
     G1[2].x=pp1[4];
     G1[2].y=pp1[5];
     G1[3].x=pp1[6];
     G1[3].y=pp1[7];
     
     hipFloatComplex T1[4];
     /* T=G1*C */
     T1[0]=hipCaddf(hipCmulf(G1[0],C[0]),hipCmulf(G1[1],C[2]));
     T1[1]=hipCaddf(hipCmulf(G1[0],C[1]),hipCmulf(G1[1],C[3]));
     T1[2]=hipCaddf(hipCmulf(G1[2],C[0]),hipCmulf(G1[3],C[2]));
     T1[3]=hipCaddf(hipCmulf(G1[2],C[1]),hipCmulf(G1[3],C[3]));

     hipFloatComplex G2[4];
     /* conjugate this */
     G2[0].x=pp2[0];
     G2[0].y=-pp2[1];
     G2[2].x=pp2[2];
     G2[2].y=-pp2[3];
     G2[1].x=pp2[4];
     G2[1].y=-pp2[5];
     G2[3].x=pp2[6];
     G2[3].y=-pp2[7];

     hipFloatComplex T2[4];
     T2[0]=hipCaddf(hipCmulf(T1[0],G2[0]),hipCmulf(T1[1],G2[2]));
     T2[1]=hipCaddf(hipCmulf(T1[0],G2[1]),hipCmulf(T1[1],G2[3]));
     T2[2]=hipCaddf(hipCmulf(T1[2],G2[0]),hipCmulf(T1[3],G2[2]));
     T2[3]=hipCaddf(hipCmulf(T1[2],G2[1]),hipCmulf(T1[3],G2[3]));
     /* update jacobian */
     /* NOTE: row major order */
     jac[m+M*8*n]=T2[0].x;
     jac[m+M*(8*n+1)]=T2[0].y;
     jac[m+M*(8*n+2)]=T2[1].x;
     jac[m+M*(8*n+3)]=T2[1].y;
     jac[m+M*(8*n+4)]=T2[2].x;
     jac[m+M*(8*n+5)]=T2[2].y;
     jac[m+M*(8*n+6)]=T2[3].x;
     jac[m+M*(8*n+7)]=T2[3].y;

    } 
   }

}


/* only use extern if calling code is C */
extern "C"
{


/* divide by singular values  Dpd[]/Sd[]  for Sd[]> eps */
void 
cudakernel_diagdiv_fl(int ThreadsPerBlock, int BlocksPerGrid, int M, float eps, float *Dpd, float *Sd) {

#ifdef CUDA_DBG
  cudaError_t error;
#endif
  kernel_diagdiv_fl<<< BlocksPerGrid, ThreadsPerBlock >>>(M, eps, Dpd, Sd);
  hipDeviceSynchronize();
#ifdef CUDA_DBG
  error = cudaGetLastError();
  if(error != cudaSuccess)
  {
    // print the CUDA error message and exit
    fprintf(stderr,"CUDA error: %s :%s: %d\n", cudaGetErrorString(error),__FILE__,__LINE__);
    exit(-1);
  }
#endif

}

/* cuda driver for calculating
  A<= A+mu I, adding mu to diagonal entries of A
  A: size MxM
  ThreadsPerBlock, BlocksPerGrid calculated to meet M
*/
void
cudakernel_diagmu_fl(int ThreadsPerBlock, int BlocksPerGrid, int M, float *A, float mu) {
#ifdef CUDA_DBG
  cudaError_t error;
#endif
  kernel_diagmu_fl<<< BlocksPerGrid, ThreadsPerBlock >>>(M, A, mu);
  hipDeviceSynchronize();
#ifdef CUDA_DBG
  error = cudaGetLastError();
  if(error != cudaSuccess)
  {
    // print the CUDA error message and exit
    fprintf(stderr,"CUDA error: %s :%s: %d\n", cudaGetErrorString(error),__FILE__,__LINE__);
    exit(-1);
  }
#endif
}


/* cuda driver for calculating f() */
/* p: params (Mx1), x: data (Nx1), other data : coh, baseline->stat mapping, Nbase, Mclusters, Nstations */
void
cudakernel_func_fl(int ThreadsPerBlock, int BlocksPerGrid, float *p, float *x, int M, int N, float *coh, short *bbh, int Nbase, int Mclus, int Nstations) {

#ifdef CUDA_DBG
  cudaError_t error;
#endif
  hipMemset(x, 0, N*sizeof(float));
//  printf("Kernel data size=%d, block=%d, thread=%d, baselines=%d\n",N,BlocksPerGrid, ThreadsPerBlock,Nbase);
  kernel_func_fl<<< BlocksPerGrid, ThreadsPerBlock >>>(Nbase,  x, coh, p, bbh, Nstations);
  hipDeviceSynchronize();
#ifdef CUDA_DBG
  error = cudaGetLastError();
  if(error != cudaSuccess)
  {
    // print the CUDA error message and exit
    fprintf(stderr,"CUDA error: %s :%s: %d\n", cudaGetErrorString(error),__FILE__,__LINE__);
    exit(-1);
  }
#endif

}

/* cuda driver for calculating jacf() */
/* p: params (Mx1), jac: jacobian (NxM), other data : coh, baseline->stat mapping, Nbase, Mclusters, Nstations */
void
cudakernel_jacf_fl(int ThreadsPerBlock_row, int  ThreadsPerBlock_col, float *p, float *jac, int M, int N, float *coh, short *bbh, int Nbase, int Mclus, int Nstations) {

#ifdef CUDA_DBG
  cudaError_t error;
#endif
  /* NOTE: use small value for ThreadsPerBlock here, like 8 */
  dim3 threadsPerBlock(16, 8);
  /* jacobian: Nbase x Nstations (proportional to N), so */
  dim3 numBlocks((Nbase+threadsPerBlock.x-1)/threadsPerBlock.x, 
               (M+threadsPerBlock.y-1)/threadsPerBlock.y);
  /* set memory of jac to zero */
  hipMemset(jac, 0, N*M*sizeof(float));
 // printf("Kernel Jax data size=%d, params=%d, block=%d,%d, thread=%d,%d, baselines=%d\n",N, M, numBlocks.x,numBlocks.y, threadsPerBlock.x, threadsPerBlock.y, Nbase);
  kernel_jacf_fl<<< numBlocks, threadsPerBlock>>>(Nbase,  M, jac, coh, p, bbh, Nstations);

  hipDeviceSynchronize();
#ifdef CUDA_DBG
  error = cudaGetLastError();
  if(error != cudaSuccess)
  {
    // print the CUDA error message and exit
    fprintf(stderr,"CUDA error: %s :%s: %d\n", cudaGetErrorString(error),__FILE__,__LINE__);
    exit(-1);
  }
#endif

}

}
